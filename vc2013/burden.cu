
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void addKernel(int *a)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	a[i] = i ^ 2;
}

extern "C" int
BurdenGPU()
{
	const int arraySize = 5000000;
	const int *a;
	int *GPU_a;

	a = (int*) malloc(arraySize*sizeof(int));
	hipMallocManaged(&GPU_a, arraySize*sizeof(int));

	int NumBlock = 50;
	int blockSize = arraySize / NumBlock;

	addKernel <<< NumBlock, blockSize >>>(GPU_a);
	hipDeviceSynchronize();

	hipMemcpy(GPU_a, a, arraySize * sizeof(int), hipMemcpyDeviceToHost);


	return 0;
}

